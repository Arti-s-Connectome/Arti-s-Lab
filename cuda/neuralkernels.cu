
#include <hip/hip_runtime.h>
extern "C"
__global__ void i2003(int n, float *a, float *b, float *c, float *d, float *I, float *u, float *v, float *vp, float *spike) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (i < n) {
		v[i] += 0.5f * (0.04f * v[i] * v[i] + 5.0f * v[i] + 140.0f - u[i] + I[i]);
		v[i] += 0.5f * (0.04f * v[i] * v[i] + 5.0f * v[i] + 140.0f - u[i] + I[i]);
		u[i] += *a * (*b * v[i] - u[i]);
		spike[i] = (v[i] >= *vp);
		u[i] = (v[i] >= *vp) * (u[i] + *d) + (v[i] < *vp) * u[i];
		v[i] = (v[i] >= *vp) * *c + (v[i] < *vp) * v[i];
	}
}